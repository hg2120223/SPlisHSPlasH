#include "hip/hip_runtime.h"
#include "magnetic_force.cuh"
#include <iostream>

std::vector<std::vector<float>> ferro(std::vector<std::vector<float>> particle_positions, float chi, float* const_ext_field) {
    int nx = 2, ny = 2, nz = 2;
    float dx = 0.0005;

    //source particles must not be empty.
    #ifndef one
    //two particles, bug free
    auto ext_source_pos = std::vector<float3>{ float3{ 0.f,-100.f, 0.f },float3{ 0.f,-101.f, 0.f } };
    auto ext_source_M = std::vector<float3>{ float3{ 0,1,0 }, float3{ 0,1,0 } };
    #else
    //one particle, singularity bug
    auto ext_source_pos = std::vector<float3>{ float3{ 0.f, 0, 0.f } };
    auto ext_source_M = std::vector<float3>{ float3{ 0,1,0 }};
    #endif
    
    
    
    auto target_pos = std::vector<float3>{};
    auto updated_target_pos = target_pos;
    updated_target_pos.reserve(nx*ny*(nz - 1));
    
    for (int i = 0; i < particle_positions.size(); i++) {
        target_pos.push_back(float3{ particle_positions[i][0], particle_positions[i][1], particle_positions[i][2] });
    }
    

    
    size_t  num_particles = target_pos.size();
    float h = dx;
    float ext_mag_h = dx;
    float volume = dx*dx*dx;
    Libo::cuda_magnet_cgOptimizer::CtorParam solver_ctor{ num_particles,h,ext_mag_h,volume,chi,const_ext_field,ext_source_pos,ext_source_M,target_pos };
    Libo::cuda_magnet_cgOptimizer a{ solver_ctor };

    a.setTargetpos(target_pos);
    Libo::cuda_magnet_cgOptimizer::setHextParams seth;
    seth.const_ext_field[0] = const_ext_field[0];
    seth.const_ext_field[1] = const_ext_field[1];
    seth.const_ext_field[2] = const_ext_field[2];
    seth.ext_source_M = ext_source_M;
    seth.ext_source_pos = ext_source_pos;
    seth.ext_mag_h = h;
    a.setHext(seth);

    a.solve(20);
    a.evaluate_fitted_near_foece();
    float * host_forcex = new float[num_particles];
    float * host_forcey = new float[num_particles];
    float * host_forcez = new float[num_particles];
    hipMemcpy(host_forcex, a.magnetic_bodyforce.dev_x, sizeof(float)*num_particles, hipMemcpyDeviceToHost);
    hipMemcpy(host_forcey, a.magnetic_bodyforce.dev_y, sizeof(float)*num_particles, hipMemcpyDeviceToHost);
    hipMemcpy(host_forcez, a.magnetic_bodyforce.dev_z, sizeof(float)*num_particles, hipMemcpyDeviceToHost);
    
    std::vector<std::vector<float>> f_ferro;
    for (int i = 0; i < target_pos.size(); i++)
    {
        f_ferro.push_back(std::vector<float>{ host_forcex[i], host_forcey[i], host_forcez[i] });
    }
    delete[] host_forcex;
    delete[] host_forcey;
    delete[] host_forcez;

    return f_ferro;
}